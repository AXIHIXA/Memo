#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <pybind11/numpy.h>

#include "util/cuax.h"


namespace cuax
{

namespace
{

//float cu(std::vector<float> & vec)
float cu(pybind11::array_t<float> & arr)
{
    auto num_items = static_cast<int>(arr.size());

    float * d_in;
    float * d_out;
    hipMalloc(&d_in, sizeof(float) * num_items);
    hipMalloc(&d_out, sizeof(float));
    hipMemcpy(d_in, arr.data(), sizeof(float) * num_items, hipMemcpyHostToDevice);

    void * d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);

    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
    hipFree(d_temp_storage);

    float res;
    hipMemcpy(&res, d_out, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

    return res;
}

}  // namespace anonymous


void init_py_module(pybind11::module_ & m)
{
    namespace py = pybind11;

//    py::bind_vector<std::vector<float>>(m, "FloatVector");

    m.def("cu", cu, py::arg("arr"), "A CUDA function.");
}

}  // namespace cuax
