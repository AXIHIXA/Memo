#include "hip/hip_runtime.h"
#include <random>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>


namespace cuax
{

__global__ void iint(
        float * __restrict__ pt,
        int * __restrict__ inside,
        int len
)
{
    // Well it proved by ncu that row-majored indexing has better throughput.
    // ncu -k regex:iint ./cmake-build-release/cumo
    int i = blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x;
    // int i = blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.x * blockDim.y + threadIdx.y;

    if (i < len)
    {
        float dx = pt[2 * i];
        float dy = pt[2 * i + 1];
        inside[i] = dx * dx + dy * dy <= 1.0f;
    }
}


class RandomPoint
{
public:
    RandomPoint() = delete;

    RandomPoint(unsigned int seed, float xMin, float xMax, float yMin, float yMax) : dx(xMin, xMax), dy(yMin, yMax) {}

    __host__ __device__ float2 operator()(unsigned long long i)
    {
        e.discard(i);
        float x = dx(e);
        float y = dy(e);
        return {x, y};
    }

private:
    thrust::default_random_engine e;
    thrust::uniform_real_distribution<float> dx;
    thrust::uniform_real_distribution<float> dy;
};


int test(int argc, char * argv[])
{
    constexpr unsigned long long kNumSamples {500000000ULL};
    unsigned int seed = std::random_device()();
    printf("seed = %u\n", seed);

    thrust::device_vector<float2> dPt(kNumSamples);
    thrust::device_vector<int> dInside(kNumSamples, 0);
    thrust::transform(
        thrust::device,
        thrust::make_counting_iterator(0ULL),
        thrust::make_counting_iterator(kNumSamples),
        dPt.begin(),
        RandomPoint(seed, -1.0f, 1.0f, -1.0f, 1.0f)
    );

    dim3 blockDim {32, 32, 1};
    unsigned int blockSize = blockDim.x * blockDim.y * blockDim.z;
    unsigned int numGrids = kNumSamples / blockSize + 1;
    dim3 gridDim {numGrids, 1, 1};

    iint<<<gridDim, blockDim>>>(reinterpret_cast<float *>(dPt.data().get()), dInside.data().get(), dInside.size());
    hipDeviceSynchronize();

    int numInside = thrust::reduce(thrust::device, dInside.begin(), dInside.end());
    printf("Monte-Carlo PI = %lf\n", static_cast<double>(numInside) / static_cast<double>(kNumSamples) * 4.0);

    return EXIT_SUCCESS;
}

}  // namespace cuax