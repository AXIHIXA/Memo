#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#include "util/add.h"


float sum(std::vector<float> & vec)
{
    auto num_items = static_cast<int>(vec.size());

    float * d_in;
    float * d_out;
    hipMalloc(&d_in, sizeof(float) * num_items);
    hipMalloc(&d_out, sizeof(float));
    hipMemcpy(d_in, vec.data(), sizeof(float) * num_items, hipMemcpyHostToDevice);

    void * d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);

    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items);
    hipFree(d_temp_storage);

    float res;
    hipMemcpy(&res, d_out, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

    return res;
}
