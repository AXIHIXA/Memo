#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <fmt/core.h>


constexpr dim3 kBlockDim {32U, 32U, 1U};

constexpr unsigned int kBlockSize {kBlockDim.x * kBlockDim.y * kBlockDim.z};

constexpr int kCenterUnrollFactor = 32;

__constant__
float2 center[kCenterUnrollFactor];

__global__
void f1(
        const float2 * __restrict__ sample,
        int sampleLen,
        int centerLen,
        float * __restrict__ res
)
{
    auto idx = static_cast<int>(blockIdx.x * (kBlockDim.x * kBlockDim.y) + threadIdx.y * blockDim.x + threadIdx.x);

    if (idx < sampleLen)
    {
        float2 r = sample[idx];

        #pragma unroll
        for (int ci = 0; ci != kCenterUnrollFactor; ++ci)
        {
            if (ci < centerLen)
            {
                float2 c = center[ci];
                atomicAdd(res + ci, (r.x - c.x) * (r.x - c.x) + (r.y - c.y) * (r.y - c.y));
            }
        }
    }
}


__global__
void f2(
        const float2 * __restrict__ sample,
        int sampleLen,
        int centerLen,
        float * __restrict__ res
)
{
    auto idx = static_cast<int>(blockIdx.x * (kBlockDim.x * kBlockDim.y) + threadIdx.y * blockDim.x + threadIdx.x);

    if (idx < sampleLen)
    {
        float2 r = sample[idx];

        #pragma unroll
        for (int ci = 0; ci != kCenterUnrollFactor; ++ci)
        {
            if (ci < centerLen)
            {
                float2 c = center[ci];
                res[sampleLen * ci + idx] = (r.x - c.x) * (r.x - c.x) + (r.y - c.y) * (r.y - c.y);
            }
        }
    }
}


int main(int argc, char * argv[])
{
    constexpr int kNumSamples = 10240010;
    constexpr int kNumCenters = 32;
    thrust::device_vector<float2> dSample(kNumSamples, {1.0f, 0.0f});
    thrust::device_vector<float2> dCenter(kNumCenters, {0.0f, 0.0f});
    dCenter.back() = {1.0f, 0.0f};
    thrust::device_vector<float> dBuffer(kNumSamples * kNumCenters, 0.0f);
    thrust::device_vector<float> dRes(kNumCenters);

    dim3 mGridDim {(kNumSamples + kBlockSize - 1) / kBlockSize, 1U, 1U};

    int numDuplication = (argc == 2) ? std::stoi(argv[1]) : 1;

    hipMemcpyToSymbol(HIP_SYMBOL(
            center),
            dCenter.data().get(),
            kNumCenters * sizeof(float2),
            0UL,
            hipMemcpyDeviceToDevice
    );

    // Warmup
    f1<<<mGridDim, kBlockDim>>>(
            dSample.data().get(),
            kNumSamples,
            kNumCenters,
            dBuffer.data().get()
    );
    hipDeviceSynchronize();

    // atomicAdd
    auto ss = std::chrono::high_resolution_clock::now();

    for (int _ = 0; _ != numDuplication; ++_)
    {
        hipMemset(dBuffer.data().get(), 0, sizeof(float) * kNumCenters);
        hipDeviceSynchronize();
        f1<<<mGridDim, kBlockDim>>>(
                dSample.data().get(),
                kNumSamples,
                kNumCenters,
                dBuffer.data().get()
        );
        hipDeviceSynchronize();
    }

    auto tic = (std::chrono::high_resolution_clock::now() - ss).count();

    std::cout << "atomicAdd "
              << static_cast<double>(tic) * 1e-6 / static_cast<double>(numDuplication)
              << " ms\n\n";

    thrust::host_vector<float> hRes = dBuffer;

    for (int i = 0; i != kNumCenters; ++i)
    {
        std::cout << hRes[i] << (i == kNumCenters - 1 ? '\n' : ' ');
    }

    // Dump global array and hipcub::DeviceReduce::Sum
    std::cout << "\n==================================\n" << '\n';
    ss = std::chrono::high_resolution_clock::now();

    thrust::device_vector<int> dOffset(kNumCenters + 1);
    thrust::sequence(thrust::device, dOffset.begin(), dOffset.end(), 0, kNumSamples);

    std::size_t tempStorageBytes;
    hipcub::DeviceSegmentedReduce::Sum(
            nullptr,
            tempStorageBytes,
            dBuffer.data().get(),
            dRes.data().get(),
            kNumCenters,
            dOffset.data().get(),
            dOffset.data().get() + 1
    );
    hipDeviceSynchronize();
    thrust::device_vector<unsigned char> dTempStorage(tempStorageBytes);

    for (int _ = 0; _ != numDuplication; ++_)
    {
        f2<<<mGridDim, kBlockDim>>>(
                dSample.data().get(),
                kNumSamples,
                kNumCenters,
                dBuffer.data().get()
        );
        hipDeviceSynchronize();

        hipcub::DeviceSegmentedReduce::Sum(
                dTempStorage.data().get(),
                tempStorageBytes,
                dBuffer.data().get(),
                dRes.data().get(),
                kNumCenters,
                dOffset.data().get(),
                dOffset.data().get() + 1
        );
        hipDeviceSynchronize();
    }

    tic = (std::chrono::high_resolution_clock::now() - ss).count();

    std::cout << "hipcub::DeviceReduce::Sum "
              << static_cast<double>(tic) * 1e-6 / static_cast<double>(numDuplication)
              << " ms\n\n";

    hRes = dRes;

    for (int i = 0; i != kNumCenters; ++i)
    {
        std::cout << hRes[i] << (i == kNumCenters - 1 ? '\n' : ' ');
    }

    return EXIT_SUCCESS;
}
