#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>


// CUDA API error checking
inline constexpr int kCudaUtilsBufferSize = 1024;

#define CUDA_CHECK(err)                                                                            \
    do {                                                                                           \
        hipError_t err_ = (err);                                                                  \
        if (err_ != hipSuccess) {                                                                 \
            char checkBuf[kCudaUtilsBufferSize] {'\0'};                                            \
            std::sprintf(checkBuf, "%s at %s:%d\n", hipGetErrorString(err_), __FILE__, __LINE__); \
            throw std::runtime_error(checkBuf);                                                    \
        }                                                                                          \
    } while (false)

#define CUDA_CHECK_LAST_ERROR()                                                                    \
    do {                                                                                           \
        hipError_t err_ = hipGetLastError();                                                     \
        if (err_ != hipSuccess) {                                                                 \
            char checkBuf[kCudaUtilsBufferSize] {'\0'};                                            \
            std::sprintf(checkBuf, "%s at %s:%d\n", hipGetErrorString(err_), __FILE__, __LINE__); \
            throw std::runtime_error(checkBuf);                                                    \
        }                                                                                          \
    } while (false)


constexpr dim3 kBlockDim {32U, 32U, 1U};

constexpr unsigned int kBlockSize {kBlockDim.x * kBlockDim.y * kBlockDim.z};

constexpr int kCenterUnrollFactor = 32;


__constant__
float2 center[kCenterUnrollFactor];


__global__
void f2(
        const float2 * __restrict__ sample,
        int sampleLen,
        int sampleLenPadded,
        int centerLen,
        float * __restrict__ res
)
{
    auto idx = static_cast<int>(blockIdx.x * (kBlockDim.x * kBlockDim.y) + threadIdx.y * blockDim.x + threadIdx.x);

    if (idx < sampleLen)
    {
        float2 r = sample[idx];

        #pragma unroll
        for (int ci = 0; ci != kCenterUnrollFactor; ++ci)
        {
            if (ci < centerLen)
            {
                float2 c = center[ci];
                res[sampleLenPadded * ci + idx] = (r.x - c.x) * (r.x - c.x) + (r.y - c.y) * (r.y - c.y);
            }
        }
    }
}


inline std::pair<int, int> padTo32k(int a)
{
    static constexpr int k32 = 32;

    if (int b = a % k32; b == 0)
    {
        return {a, 0};
    }
    else
    {
        return {a + k32 - b, b};
    }
}


int main(int argc, char * argv[])
{
    int numDuplication = (argc == 2) ? std::stoi(argv[1]) : 1;

    static constexpr int kNumSamplesInit = 10244321;
    static constexpr int kNumCentersInit = 8196;

    int numSamples = kNumSamplesInit;
    int numCenters = kNumCentersInit;
    int numCentersLastBatch = numCenters % kCenterUnrollFactor;

    thrust::device_vector<float2> dSample(numSamples, {1.0f, 0.0f});
    thrust::device_vector<float2> dCenter(numCenters, {0.0f, 0.0f});
    dCenter[numCenters - 4] = {1.0f, 0.0f};
    dCenter[numCenters - 3] = {1.0f, 1.0f};
    dCenter[numCenters - 2] = {0.0f, 1.0f};
    dCenter[numCenters - 1] = {1.0f, 0.0f};

    // Pad numSamples to multiple of 32
    // (32 * sizeof(float) == 128, L1 cache line granularity)
    // for aligned & coalesced memory access pattern.
    auto [numSamplesPadded, remainder] = padTo32k(numSamples);
    // int numSamplesPadded = numSamples;  // Test for non-aligned pattern. Slower!
    thrust::device_vector<float> dBuffer(numSamplesPadded * kCenterUnrollFactor, 0.0f);
    dim3 mGridDim {(numSamples + kBlockSize - 1) / kBlockSize, 1U, 1U};

    // Test
    auto ss = std::chrono::high_resolution_clock::now();

    thrust::device_vector<int> dBeginOffset(numCenters);
    thrust::device_vector<int> dEndOffset(numCenters);
    thrust::device_vector<float> dResult(numCenters);
    thrust::sequence(thrust::device, dBeginOffset.begin(), dBeginOffset.end(), 0, numSamplesPadded);
    thrust::sequence(thrust::device, dEndOffset.begin(), dEndOffset.end(), numSamples, numSamplesPadded);

    std::size_t tempStorageBytes;
    CUDA_CHECK(
            hipcub::DeviceSegmentedReduce::Sum(
                    nullptr,
                    tempStorageBytes,
                    dBuffer.data().get(),
                    dResult.data().get(),
                    numCenters,
                    dBeginOffset.data().get(),
                    dEndOffset.data().get()
            )
    );
    CUDA_CHECK(hipDeviceSynchronize());
    thrust::device_vector<unsigned char> dTempStorage(tempStorageBytes);

    CUDA_CHECK(
            hipMemsetAsync(
                    dBuffer.data().get(),
                    0,
                    sizeof(float) * dBuffer.size(),
                    nullptr
            )
    );

    for (int _ = 0; _ != numDuplication; ++_)
    {
        for (int ci = 0; ci < numCenters; ci += kCenterUnrollFactor)
        {
            int numCentersThisBatch =
                    (numCentersLastBatch != 0 and numCenters <= ci + kCenterUnrollFactor) ?
                    numCentersLastBatch :
                    kCenterUnrollFactor;

            CUDA_CHECK(
                    hipMemcpyToSymbolAsync(HIP_SYMBOL(
                            center),
                            dCenter.data().get() + ci,
                            numCentersThisBatch * sizeof(float2),
                            0UL,
                            hipMemcpyDeviceToDevice,
                            nullptr
                    )
            );

            f2<<<mGridDim, kBlockDim, 0U, nullptr>>>(
                    dSample.data().get(),
                    numSamples,
                    numSamplesPadded,
                    numCentersThisBatch,
                    dBuffer.data().get()
            );
            CUDA_CHECK_LAST_ERROR();

            CUDA_CHECK(
                    hipcub::DeviceSegmentedReduce::Sum(
                            dTempStorage.data().get(),
                            tempStorageBytes,
                            dBuffer.data().get(),
                            dResult.data().get() + ci,
                            numCentersThisBatch,
                            dBeginOffset.data().get(),
                            dEndOffset.data().get(),
                            nullptr
                    )
            );

            CUDA_CHECK(
                    hipMemsetAsync(
                            dBuffer.data().get(),
                            0,
                            sizeof(float) * dBuffer.size(),
                            nullptr
                    )
            );
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    auto tic = (std::chrono::high_resolution_clock::now() - ss).count();

    std::cout << "Before Unrolling "
              << static_cast<double>(tic) * 1e-6 / static_cast<double>(numDuplication)
              << " ms\n\n";

    if (numDuplication == 1)
    {
        thrust::host_vector<float> hRes = dResult;

        for (int i = 0; i != numCenters; ++i)
        {
            std::cout << "sum @ center[" << i << "] = " << hRes[i] << '\n';
        }
    }

    return EXIT_SUCCESS;
}


/*
$ ./cmake-build-release/exe 40

Before Unrolling 1933.27 ms
Before Unrolling 1967.84 ms
*/
