#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>


constexpr dim3 kBlockDim {32U, 32U, 1U};

constexpr unsigned int kBlockSize {kBlockDim.x * kBlockDim.y * kBlockDim.z};

constexpr int kCenterUnrollFactor = 32;


__constant__
float2 center[kCenterUnrollFactor];


__global__
void f2(
        const float2 * __restrict__ sample,
        int sampleLen,
        int sampleLenPadded,
        int centerLen,
        float * __restrict__ res
)
{
    auto idx = static_cast<int>(blockIdx.x * (4 * kBlockDim.x * kBlockDim.y) + threadIdx.y * blockDim.x + threadIdx.x);

    if (idx + 3 * kBlockDim.x * kBlockDim.y < sampleLen)
    {
        float2 r0 = sample[idx];
        float2 r1 = sample[idx +     kBlockDim.x * kBlockDim.y];
        float2 r2 = sample[idx + 2 * kBlockDim.x * kBlockDim.y];
        float2 r3 = sample[idx + 3 * kBlockDim.x * kBlockDim.y];

        auto resIdx = static_cast<int>(blockIdx.x * kBlockDim.x * kBlockDim.y + threadIdx.y * blockDim.x + threadIdx.x);

        #pragma unroll
        for (int ci = 0; ci != kCenterUnrollFactor; ++ci)
        {
            if (ci < centerLen)
            {
                float2 c = center[ci];
                float nr0c = (r0.x - c.x) * (r0.x - c.x) + (r0.y - c.y) * (r0.y - c.y);
                float nr1c = (r1.x - c.x) * (r1.x - c.x) + (r1.y - c.y) * (r1.y - c.y);
                float nr2c = (r2.x - c.x) * (r2.x - c.x) + (r2.y - c.y) * (r2.y - c.y);
                float nr3c = (r3.x - c.x) * (r3.x - c.x) + (r3.y - c.y) * (r3.y - c.y);
                res[sampleLenPadded * ci + resIdx] = nr0c + nr1c + nr2c + nr3c;
            }
        }
    }
}


std::pair<int, int> padTo32k(int a)
{
    static constexpr int k32 = 32;

    if (int b = a % k32; b == 0)
    {
        return {a, 0};
    }
    else
    {
        return {a + k32 - b, b};
    }
}


int main(int argc, char * argv[])
{
    int numDuplication = (argc == 2) ? std::stoi(argv[1]) : 1;

    static constexpr int kNumSamplesInit = 10240000;
    static constexpr int kNumCentersInit = 8196;

    int numSamples = kNumSamplesInit;
    int numCenters = kNumCentersInit;
    int numCentersLastBatch = numCenters % kCenterUnrollFactor;

    thrust::device_vector<float2> dSample(numSamples, {1.0f, 0.0f});
    thrust::device_vector<float2> dCenter(numCenters, {0.0f, 0.0f});
    dCenter[numCenters - 4] = {1.0f, 0.0f};
    dCenter[numCenters - 3] = {1.0f, 1.0f};
    dCenter[numCenters - 2] = {0.0f, 1.0f};
    dCenter[numCenters - 1] = {1.0f, 0.0f};

    // Pad numSamples to multiple of 32
    // (32 * sizeof(float) == 128, L1 cache line granularity)
    // for aligned & coalesced memory access pattern.
    auto [numSamplesPadded, remainder] = padTo32k(numSamples >> 2U);
    // int numSamplesPadded = numSamples;  // Test for non-aligned pattern. Slower!
    thrust::device_vector<float> dBuffer(numSamplesPadded * kCenterUnrollFactor, 0.0f);

    dim3 mGridDim {(numSamples + 2 * kBlockSize - 1) / (2 * kBlockSize), 1U, 1U};

    // Test
    auto ss = std::chrono::high_resolution_clock::now();

    thrust::device_vector<int> dBeginOffset(numCenters);
    thrust::device_vector<int> dEndOffset(numCenters);
    thrust::device_vector<float> dResult(numCenters);
    thrust::sequence(thrust::device, dBeginOffset.begin(), dBeginOffset.end(), 0, numSamplesPadded);
    thrust::sequence(thrust::device, dEndOffset.begin(), dEndOffset.end(), numSamples >> 2U, numSamplesPadded);

    std::size_t tempStorageBytes;
    hipcub::DeviceSegmentedReduce::Sum(
            nullptr,
            tempStorageBytes,
            dBuffer.data().get(),
            dResult.data().get(),
            numCenters,
            dBeginOffset.data().get(),
            dEndOffset.data().get()
    );
    hipDeviceSynchronize();
    thrust::device_vector<unsigned char> dTempStorage(tempStorageBytes);

    hipMemsetAsync(dBuffer.data().get(), 0, sizeof(float) * dBuffer.size(), nullptr);

    for (int _ = 0; _ != numDuplication; ++_)
    {
        for (int ci = 0; ci < numCenters; ci += kCenterUnrollFactor)
        {
            int numCentersThisBatch =
                    (numCentersLastBatch != 0 and numCenters <= ci + kCenterUnrollFactor) ?
                    numCentersLastBatch :
                    kCenterUnrollFactor;

            hipMemcpyToSymbolAsync(HIP_SYMBOL(
                    center),
                    dCenter.data().get() + ci,
                    numCentersThisBatch * sizeof(float2),
                    0UL,
                    hipMemcpyDeviceToDevice,
                    nullptr
            );

            f2<<<mGridDim, kBlockDim, 0U, nullptr>>>(
                    dSample.data().get(),
                    numSamples,
                    numSamplesPadded,
                    numCentersThisBatch,
                    dBuffer.data().get()
            );

            hipcub::DeviceSegmentedReduce::Sum(
                    dTempStorage.data().get(),
                    tempStorageBytes,
                    dBuffer.data().get(),
                    dResult.data().get() + ci,
                    numCentersThisBatch,
                    dBeginOffset.data().get(),
                    dEndOffset.data().get(),
                    nullptr
            );

            hipMemsetAsync(dBuffer.data().get(), 0, sizeof(float) * dBuffer.size(), nullptr);
        }
    }

    hipDeviceSynchronize();
    auto tic = (std::chrono::high_resolution_clock::now() - ss).count();

    std::cout << "After Unrolling "
              << static_cast<double>(tic) * 1e-6 / static_cast<double>(numDuplication)
              << " ms\n\n";

    if (numDuplication == 1)
    {
        thrust::host_vector<float> hRes = dResult;

        for (int i = 0; i != numCenters; ++i)
        {
            std::cout << "sum @ center[" << i << "] = " << hRes[i] << '\n';
        }
    }

    return EXIT_SUCCESS;
}


/*
$ ./cmake-build-release/exe 40

After Unrolling 587.185 ms
After Unrolling 551.579 ms
*/
