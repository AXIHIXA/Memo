#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>


constexpr dim3 kBlockDim {32U, 32U, 1U};

constexpr unsigned int kBlockSize {kBlockDim.x * kBlockDim.y * kBlockDim.z};

constexpr int kCenterUnrollFactor = 32;


__constant__
float2 center[kCenterUnrollFactor];


__global__
void f2(
        const float2 * __restrict__ sample,
        int sampleLen,
        int sampleLenPadded,
        int centerLen,
        float * __restrict__ res
)
{
    auto idx = static_cast<int>(blockIdx.x * (kBlockDim.x * kBlockDim.y << 2U) + threadIdx.y * blockDim.x + threadIdx.x);

    if (idx < sampleLen)
    {
        float2 r = sample[idx];

        float2 r2;
        if (idx + kBlockSize < sampleLen)
            r2 = sample[idx + kBlockSize];
        float2 r3;
        if (idx + (kBlockSize << 1U) < sampleLen)
            r3 = sample[idx + (kBlockSize << 1U)];
        float2 r4;
        if (idx + (kBlockSize * 3) < sampleLen)
            r4 = sample[idx + (kBlockSize * 3)];

        #pragma unroll
        for (int ci = 0; ci != kCenterUnrollFactor; ++ci)
        {
            if (ci < centerLen)
            {
                float2 c = center[ci];
                res[sampleLenPadded * ci + idx] = (r.x - c.x) * (r.x - c.x) + (r.y - c.y) * (r.y - c.y);

                if (idx + kBlockSize < sampleLen)
                    res[sampleLenPadded * ci + idx + kBlockSize] =
                            (r2.x - c.x) * (r2.x - c.x) + (r2.y - c.y) * (r2.y - c.y);
                if (idx + (kBlockSize << 1U) < sampleLen)
                    res[sampleLenPadded * ci + idx + (kBlockSize << 1U)] =
                            (r3.x - c.x) * (r3.x - c.x) + (r3.y - c.y) * (r3.y - c.y);
                if (idx + (kBlockSize * 3) < sampleLen)
                    res[sampleLenPadded * ci + idx + (kBlockSize * 3)] =
                            (r4.x - c.x) * (r4.x - c.x) + (r4.y - c.y) * (r4.y - c.y);
            }
        }
    }
}


std::pair<int, int> padTo128k(int a)
{
    static constexpr int k128 = 128;

    if (int b = a % k128; b == 0)
    {
        return {a, 0};
    }
    else
    {
        return {a + k128 - b, b};
    }
}


int main(int argc, char * argv[])
{
    int numDuplication = (argc == 2) ? std::stoi(argv[1]) : 1;

    static constexpr int kNumSamplesInit = 10240010;
    static constexpr int kNumCentersInit = 8196;

    int numSamples = kNumSamplesInit;
    int numCenters = kNumCentersInit;
    int numCentersLastBatch = numCenters % kCenterUnrollFactor;

    thrust::device_vector<float2> dSample(numSamples, {1.0f, 0.0f});
    thrust::device_vector<float2> dCenter(numCenters, {0.0f, 0.0f});
    dCenter[numCenters - 4] = {1.0f, 0.0f};
    dCenter[numCenters - 3] = {1.0f, 0.0f};
    dCenter[numCenters - 2] = {0.0f, 1.0f};
    dCenter[numCenters - 1] = {1.0f, 0.0f};

    // Pad numSamples to multiple of 128 for aligned & coalesced memory access pattern.
    auto [numSamplesPadded, remainder] = padTo128k(numSamples);
    thrust::device_vector<float> dBuffer(numSamplesPadded * kCenterUnrollFactor, 0.0f);

    dim3 mGridDim {(numSamples + (kBlockSize << 2U) - 1) / (kBlockSize << 2U), 1U, 1U};

    // Warmup
    if (1 < numDuplication)
    {
        f2<<<mGridDim, kBlockDim>>>(
                dSample.data().get(),
                numSamples,
                numSamplesPadded,
                numCenters,
                dBuffer.data().get()
        );
        hipDeviceSynchronize();
    }

    // atomicAdd
    // Dump global array and hipcub::DeviceReduce::Sum
    auto ss = std::chrono::high_resolution_clock::now();

    thrust::device_vector<int> dBeginOffset(numCenters);
    thrust::device_vector<int> dEndOffset(numCenters);
    thrust::device_vector<float> dResult(numCenters);
    thrust::sequence(thrust::device, dBeginOffset.begin(), dBeginOffset.end(), 0, numSamplesPadded);
    thrust::sequence(thrust::device, dEndOffset.begin(), dEndOffset.end(), numSamples, numSamplesPadded);

    std::size_t tempStorageBytes;
    hipcub::DeviceSegmentedReduce::Sum(
            nullptr,
            tempStorageBytes,
            dBuffer.data().get(),
            dResult.data().get(),
            numCenters,
            dBeginOffset.data().get(),
            dEndOffset.data().get()
    );
    hipDeviceSynchronize();
    thrust::device_vector<unsigned char> dTempStorage(tempStorageBytes);

    for (int _ = 0; _ != numDuplication; ++_)
    {
        for (int ci = 0; ci < numCenters; ci += kCenterUnrollFactor)
        {
            int numCentersThisBatch =
                    (numCentersLastBatch != 0 and numCenters < ci + kCenterUnrollFactor) ?
                    numCentersLastBatch :
                    kCenterUnrollFactor;

            hipMemcpyToSymbol(HIP_SYMBOL(
                    center),
                    dCenter.data().get() + ci,
                    numCentersThisBatch * sizeof(float2),
                    0UL,
                    hipMemcpyDeviceToDevice
            );

            f2<<<mGridDim, kBlockDim>>>(
                    dSample.data().get(),
                    numSamples,
                    numSamplesPadded,
                    numCentersThisBatch,
                    dBuffer.data().get()
            );
            hipDeviceSynchronize();

            hipcub::DeviceSegmentedReduce::Sum(
                    dTempStorage.data().get(),
                    tempStorageBytes,
                    dBuffer.data().get(),
                    dResult.data().get() + ci,
                    numCentersThisBatch,
                    dBeginOffset.data().get(),
                    dEndOffset.data().get()
            );
            hipDeviceSynchronize();
        }
    }

    auto tic = (std::chrono::high_resolution_clock::now() - ss).count();

    std::cout << "ConstantCenter + hipcub::DeviceSegmentedReduce::Sum "
              << static_cast<double>(tic) * 1e-6 / static_cast<double>(numDuplication)
              << " ms\n\n";

    if (numDuplication == 1)
    {
        thrust::host_vector<float> hRes = dResult;

        for (int i = 0; i != numCenters; ++i)
        {
            std::cout << "sum @ center[" << i << "] = " << hRes[i] << '\n';
        }
    }

    return EXIT_SUCCESS;
}


/*
$ ./cmake-build-release/exe 100
ConstantCenter + hipcub::DeviceSegmentedReduce::Sum 7.02152 ms

1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 1.024e+07 0 2.048e+07 0
*/
