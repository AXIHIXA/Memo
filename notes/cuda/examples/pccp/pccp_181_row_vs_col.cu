#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>


constexpr dim3 kBlockDim {32U, 32U, 1U};
constexpr unsigned int kBlockSize {kBlockDim.x * kBlockDim.y * kBlockDim.z};


__global__
void readColumnStoreRow(float * __restrict__ dst, const float * __restrict__ src, int n)
{
    auto idx = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
    auto idy = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

    if (idx < n and idy < n)
    {
        dst[idy * n + idx] = src[idx * n + idy];
    }
}


__global__
void readRowStoreColumn(float * __restrict__ dst, const float * __restrict__ src, int n)
{
    auto idx = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
    auto idy = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

    if (idx < n and idy < n)
    {
        dst[idx * n + idy] = src[idy * n + idx];
    }
}


__global__
void warmup(float * __restrict__ dst, const float * __restrict__ src, int n)
{
    auto idx = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
    auto idy = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);

    if (idx < n and idy < n)
    {
        dst[idy * n + idx] = src[idy * n + idx];
    }
}


int main(int argc, char * argv[])
{
    constexpr int kNumSamples = 20480000;
    int numDuplication = (argc == 2) ? 1000 : 1;
    int nn = 32 * 32;

    thrust::device_vector<float> dIn(kNumSamples);
    thrust::device_vector<float> dOut(kNumSamples);

    dim3 mGridDim {32U, 32U, 1U};

    // Warmup
    warmup<<<mGridDim, kBlockDim>>>(
            dOut.begin().base().get(),
            dIn.begin().base().get(),
            nn
    );

    hipDeviceSynchronize();

    // Acutal Tests
    for (int _ = 0; _ != numDuplication; ++_)
    {
        readColumnStoreRow<<<mGridDim, kBlockDim>>>(
                dOut.begin().base().get(),
                dIn.begin().base().get(),
                nn
        );
    }

    hipDeviceSynchronize();

    for (int _ = 0; _ != numDuplication; ++_)
    {
        readRowStoreColumn<<<mGridDim, kBlockDim>>>(
                dOut.begin().base().get(),
                dIn.begin().base().get(),
                nn
        );
    }

    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}


/*
$ ncu -k regex:read --metrics l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second,l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second,smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct,smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct ./cmake-build-release/exe

  readColumnStoreRow(float *, const float *, int)
    Section: Command line profiler metrics
    ---------------------------------------------------------------------- --------------- ------------------------------
    l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second                   Gbyte/second                         887.87
    l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second                   Gbyte/second                         110.98
    smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct                        %                          12.50
    smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct                        %                            100
    ---------------------------------------------------------------------- --------------- ------------------------------

  readRowStoreColumn(float *, const float *, int)
    Section: Command line profiler metrics
    ---------------------------------------------------------------------- --------------- ------------------------------
    l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second                   Gbyte/second                          70.24
    l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second                   Gbyte/second                         561.94
    smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct                        %                            100
    smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct                        %                          12.50
    ---------------------------------------------------------------------- --------------- ------------------------------
*/

/*
$ nvprof ./cmake-build-release/exe adfhklahfklasd

            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   59.56%  59.372ms      1000  59.372us  56.255us  800.12us  readRowStoreColumn(float*, float const *, int)
                   40.10%  39.970ms      1000  39.969us  35.936us  40.927us  readColumnStoreRow(float*, float const *, int)
*/
