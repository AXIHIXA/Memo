#include "hip/hip_runtime.h"
#include <random>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>


namespace
{

constexpr int kVecLen = 640000000;

}  // namespace anomynous


int dumpRandomVector()
{
    unsigned int seed = std::random_device()();
    std::cout << "seed = " << seed << '\n';
    std::default_random_engine e(seed);
    std::uniform_real_distribution<float> g(-1.0f, 1.0f);

    std::vector<float> hVec(kVecLen, 0.0f);

    for (int i = 0LL; i != kVecLen; ++i)
    {
        hVec[i] = g(e);
    }

    if (std::FILE * fp = std::fopen("var/vec.bin", "wb"); fp)
    {
        std::fwrite(hVec.begin().base(), sizeof(float) * hVec.size(), 0UL, fp);
        std::fclose(fp);
    }
    else
    {
        std::fclose(fp);
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}


struct ss
{
    float x;
    float y;
};


__global__
void arrayOfStructure(
    const ss * __restrict__ aos,
    ss * __restrict__ res,
    int len
)
{
    auto i = static_cast<int>(blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x);

    if (i < len)
    {
        ss tmp = aos[i];
        tmp.x += 0.0001f;
        tmp.y += 0.0001f;
        res[i]= tmp;
    }
}


__global__
void structureOfArray(
        const float * __restrict__ x,
        const float * __restrict__ y,
        float * __restrict__ resX,
        float * __restrict__ resY,
        int len
)
{
    auto i = static_cast<int>(blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x);

    if (i < len)
    {
        float tmpX = x[i];
        float tmpY = y[i];
        tmpX += 0.0001f;
        tmpY += 0.0001f;
        resX[i] = tmpX;
        resY[i] = tmpY;
    }
}


int main(int argc, char * argv[])
{
    std::vector<float> hVec(kVecLen);

    if (std::FILE * fp = std::fopen("var/vec.bin", "rb"))
    {
        [[maybe_unused]] std::size_t numObjectsRead = std::fread(hVec.begin().base(), sizeof(float), kVecLen, fp);
        std::fclose(fp);
    }
    else
    {
        std::fclose(fp);
        return EXIT_FAILURE;
    }

    thrust::device_vector<float> dVec = hVec;
    thrust::device_vector<float> dRes(kVecLen);

    auto dAos = reinterpret_cast<ss *>(dVec.begin().base().get());
    auto dAosRes = reinterpret_cast<ss *>(dRes.begin().base().get());

    auto dX = dVec.begin().base().get();
    auto dY = dX + (kVecLen >> 1U);
    auto dXRes = dRes.begin().base().get();
    auto dYRes = dXRes + (kVecLen >> 1U);

    if (argc != 2)
    {
        return EXIT_FAILURE;
    }

    if (std::string(argv[1]) == "aos")
    {
        arrayOfStructure<<<dim3((kVecLen >> 1U) / 1024 + 1, 1, 1), dim3(32, 32, 1)>>>(
            dAos, dAosRes, kVecLen >> 1U
        );
    }
    else
    {
        structureOfArray<<<dim3((kVecLen >> 1U) / 1024 + 1, 1, 1), dim3(32, 32, 1)>>>(
            dX, dY, dXRes, dYRes, kVecLen >> 1U
        );
    }

    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}

/*
$ ncu -k regex:arrayOfStructure --metrics l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second,l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second,smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct,smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct ./cmake-build-release/cumo aos

  arrayOfStructure(const ss *, ss *, int)
    Section: Command line profiler metrics
    ---------------------------------------------------------------------- --------------- ------------------------------
    l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second                   Gbyte/second                         544.67
    l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second                   Gbyte/second                         544.67
    smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct                        %                             50
    smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct                        %                             50
    ---------------------------------------------------------------------- --------------- ------------------------------


$ ncu -k regex:structureOfArray --metrics l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second,l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second,smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct,smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct ./cmake-build-release/cumo soa

  structureOfArray(const float *, const float *, float *, float *, int)
    Section: Command line profiler metrics
    ---------------------------------------------------------------------- --------------- ------------------------------
    l1tex__t_bytes_pipe_lsu_mem_global_op_ld.sum.per_second                   Gbyte/second                         271.27
    l1tex__t_bytes_pipe_lsu_mem_global_op_st.sum.per_second                   Gbyte/second                         271.27
    smsp__sass_average_data_bytes_per_sector_mem_global_op_ld.pct                        %                            100
    smsp__sass_average_data_bytes_per_sector_mem_global_op_st.pct                        %                            100
    ---------------------------------------------------------------------- --------------- ------------------------------
*/
